#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <vector>
#include <unordered_map>
#include <algorithm>
#include "errors.h"

#define MAX_VERTICES 5000000
#define MAX_EDGES 5000000
#define MAX_DEG 1024
#define MAX_DEPTH K

#define BLOCK_SIZE 32
#define MAX_NUM_BLOCKS 128
#define GROUP_SIZE 8
#define GROUPS_PER_BLOCK (int)(BLOCK_SIZE / GROUP_SIZE)

#define MOD 1000000000

std::vector<std::pair<uint, uint>> edges;
std::unordered_map<uint, uint> degree;

std::unordered_map<uint, uint> id_to_number;
std::unordered_map<uint, uint> number_to_id;

__global__ void kcliques(std::pair<uint, uint>* edges, std::pair<int, int>* intervals, int N, unsigned int* intersect, uint* stackVertex, int* stackDepth, int* cliques, int K, unsigned int* inducedSubgraph, int maxDeg, int maxStackSize) {
    uint mask = ((~(uint)0) >> (32 - GROUP_SIZE));
    int groupInWarp = (threadIdx.x % 32) / GROUP_SIZE;
    mask = mask << (groupInWarp * GROUP_SIZE);

    __shared__ int maxStackTop[GROUPS_PER_BLOCK];

    int stackTop;
    int pref;

    int part = N / gridDim.x;
    int rest = N % gridDim.x;

    int firstVertexIncl = blockIdx.x * part + min(blockIdx.x, rest);
    int lastVertexExcl = (blockIdx.x + 1) * part + min(blockIdx.x + 1, rest);

    for (int v = firstVertexIncl; v < lastVertexExcl; ++v) { // Rozważamy graf indukowany zbiorem sąsiadów v
        int graphSize = intervals[v].second - intervals[v].first;
        int graphPart = graphSize / blockDim.x;
        int graphRest = graphSize % blockDim.x;

        int firstNeighbourIncl = threadIdx.x * graphPart + min(threadIdx.x, graphRest);
        int lastNeighbourExcl = (threadIdx.x + 1) * graphPart + min(threadIdx.x + 1, graphRest);

        int codeSize = (graphSize + 31) / 32; // Na tylu liczbach kodujemy wiersz macierzy sąsiedztwa

        for (int i = firstNeighbourIncl; i < lastNeighbourExcl; ++i) {
            uint u = edges[i + intervals[v].first].second; // Kopiujemy listę sąsiedztwa tego sąsiada
            // i = numer odpowiadający u w indukowanym podgrafie

            // Czyścimy wiersz odpowiadający u w inducedSubgraph
            for (int j = 0; j < codeSize; ++j) {
                inducedSubgraph[blockIdx.x * maxDeg * (maxDeg / 32) + i * (maxDeg / 32) + j] = 0;
            }

            for (int j = intervals[u].first; j < intervals[u].second; j++) {
                uint w = edges[j].second;
                // Sprawdzamy, czy w jest w indukowanym podgrafie
                int left = 0, right = graphSize - 1, mid;
                while (left < right) {
                    mid = (left + right) / 2;
                    if (edges[mid + intervals[v].first].second < w) {
                        left = mid + 1;
                    }
                    else {
                        right = mid;
                    }
                }
                if (edges[left + intervals[v].first].second == w) {
                    // left = numer odpowiadający w w indukowanym podgrafie
                    int number = left / 32;
                    int bit = left % 32;
                    inducedSubgraph[blockIdx.x * maxDeg * (maxDeg / 32) + i * (maxDeg / 32) + number] |= (1 << bit);
                }
            }
        }

        // Wykonujemy DFS

        for (int i = firstNeighbourIncl; i < lastNeighbourExcl; ++i) {
            // i-ty sąsiad trafia na stos grupy i % GROUPS_PER_BLOCK na pozycję i / GROUPS_PER_BLOCK
            stackVertex[blockIdx.x * GROUPS_PER_BLOCK * maxStackSize + (i % GROUPS_PER_BLOCK) * maxStackSize + (i / GROUPS_PER_BLOCK)] = i;
            stackDepth[blockIdx.x * GROUPS_PER_BLOCK * maxStackSize + (i % GROUPS_PER_BLOCK) * maxStackSize + (i / GROUPS_PER_BLOCK)] = 1;
        }

        int codePart = codeSize / GROUP_SIZE;
        int codeRest = codeSize % GROUP_SIZE;

        int threadInGroup = threadIdx.x % GROUP_SIZE; // Którym jestem wątkiem w swojej grupie
        int groupId = threadIdx.x / GROUP_SIZE; // Numer mojej grupy

        int firstIntersectionIncl = threadInGroup * codePart + min(threadInGroup, codeRest);
        int lastIntersectionExcl = (threadInGroup + 1) * codePart + min(threadInGroup + 1, codeRest);


        // Więcej jednynek niż graphSize. Czy to nie przeszkadza?
        for (int i = firstIntersectionIncl; i < lastIntersectionExcl; ++i) {
            intersect[blockIdx.x * GROUPS_PER_BLOCK * MAX_DEPTH * (maxDeg / 32) + groupId * MAX_DEPTH * (maxDeg / 32) + 0 * (maxDeg / 32) + i] = ~0;
        }

        stackTop = (graphSize / GROUPS_PER_BLOCK) + (graphSize % GROUPS_PER_BLOCK > groupId ? 1 : 0) - 1;

        if (threadIdx.x == 0) {
            cliques[0 * gridDim.x * GROUPS_PER_BLOCK + blockIdx.x * GROUPS_PER_BLOCK + groupId]++;
            cliques[0 * gridDim.x * GROUPS_PER_BLOCK + blockIdx.x * GROUPS_PER_BLOCK + groupId] %= MOD;
        }
        if (threadInGroup == 0) {
            maxStackTop[groupId] = stackTop;
            cliques[1 * gridDim.x * GROUPS_PER_BLOCK + blockIdx.x * GROUPS_PER_BLOCK + groupId] += (graphSize / GROUPS_PER_BLOCK) + (graphSize % GROUPS_PER_BLOCK > groupId ? 1 : 0); // Odpowiada wszystkim tym wrzuconym na stos wierzchołkom
            cliques[1 * gridDim.x * GROUPS_PER_BLOCK + blockIdx.x * GROUPS_PER_BLOCK + groupId] %= MOD;
        }
        __syncthreads();

        for (int i = 1; i < GROUPS_PER_BLOCK; i *= 2) {
            if (threadInGroup == 0) {
                maxStackTop[groupId] = max(maxStackTop[(groupId + i) % GROUPS_PER_BLOCK], maxStackTop[groupId]);
            }
            __syncthreads();
        }

        while(maxStackTop[groupId] >= 0) {

            if (stackTop >= 0) {
                uint u = stackVertex[blockIdx.x * GROUPS_PER_BLOCK * maxStackSize + groupId * maxStackSize + stackTop];
                int depth = stackDepth[blockIdx.x * GROUPS_PER_BLOCK * maxStackSize + groupId * maxStackSize + stackTop];

                int children = 0;
                for (int i = firstIntersectionIncl; i < lastIntersectionExcl; ++i) {
                    intersect[blockIdx.x * GROUPS_PER_BLOCK * MAX_DEPTH * (maxDeg / 32) + groupId * MAX_DEPTH * (maxDeg / 32) + depth * (maxDeg / 32) + i] = intersect[blockIdx.x * GROUPS_PER_BLOCK * MAX_DEPTH * (maxDeg / 32) + groupId * MAX_DEPTH * (maxDeg / 32) + (depth - 1) * (maxDeg / 32) + i] & inducedSubgraph[blockIdx.x * maxDeg * (maxDeg / 32) + u * (maxDeg / 32) + i];
                    for (int bit = 0; bit < 32; bit++) {
                        children += ((intersect[blockIdx.x * GROUPS_PER_BLOCK * MAX_DEPTH * (maxDeg / 32) + groupId * MAX_DEPTH * (maxDeg / 32) + depth * (maxDeg / 32) + i] >> bit) & 1);
                    }
                }

                pref = children;

                for (int i = 1; i < GROUP_SIZE; i *= 2) {
                    int tmp = __shfl_sync(mask, pref, threadInGroup - i, GROUP_SIZE);
                    pref += (threadInGroup >= i ? tmp : 0);
                }

                if (depth + 1 < K - 1) {
                    int pos = stackTop + pref - children;
                    for (int i = firstIntersectionIncl; i < lastIntersectionExcl; ++i) {
                        for (int bit = 0; bit < 32; bit++) {
                            if ((intersect[blockIdx.x * GROUPS_PER_BLOCK * MAX_DEPTH * (maxDeg / 32) + groupId * MAX_DEPTH * (maxDeg / 32) + depth * (maxDeg / 32) + i] >> bit) & 1 == 1) {
                                stackVertex[blockIdx.x * GROUPS_PER_BLOCK * maxStackSize + groupId * maxStackSize + pos] = i * 32 + bit;
                                stackDepth[blockIdx.x * GROUPS_PER_BLOCK * maxStackSize + groupId * maxStackSize + pos] = depth + 1;
                                pos++;
                            }
                        }
                    }
                }

                if (threadInGroup == GROUP_SIZE - 1) {
                    cliques[(depth + 1) * gridDim.x * GROUPS_PER_BLOCK + blockIdx.x * GROUPS_PER_BLOCK + groupId] += pref;
                    cliques[(depth + 1) * gridDim.x * GROUPS_PER_BLOCK + blockIdx.x * GROUPS_PER_BLOCK + groupId] %= MOD;
                    stackTop = (depth + 1 < K - 1 ? stackTop + pref - 1 : stackTop - 1);
                }

                stackTop = __shfl_sync(mask, stackTop, groupId * GROUP_SIZE + (GROUP_SIZE - 1), GROUP_SIZE);
            }

            if (threadInGroup == 0) {
                maxStackTop[groupId] = stackTop;
            }

            __syncthreads();

            for (int i = 1; i < GROUPS_PER_BLOCK; i *= 2) {
                if (threadInGroup == 0) {
                    maxStackTop[groupId] = max(maxStackTop[(groupId + i) % GROUPS_PER_BLOCK], maxStackTop[groupId]);
                }
                __syncthreads();
            }
        }
    }
}

__global__ void reduce(int* cliques, int columns) {
    __shared__ int row[MAX_NUM_BLOCKS * GROUPS_PER_BLOCK];

    row[threadIdx.x] = cliques[blockIdx.x * columns + threadIdx.x];

    __syncthreads();

    for (int i = 1; i < columns; i *= 2) {
        if (threadIdx.x % (2 * i) == 0 && threadIdx.x + i < columns) {
            row[threadIdx.x] += row[threadIdx.x + i];
            row[threadIdx.x] %= MOD;
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        cliques[blockIdx.x] = row[0];
    }
}

int main(int argc, char* argv[]) {

    if (32 % GROUP_SIZE != 0) {
        std::cerr << "Warp size must be a multiple of GROUP_SIZE";
        return 1;
    }
    if (BLOCK_SIZE % GROUP_SIZE != 0) {
        std::cerr << "BLOCK_SIZE must be a multiple of GROUP_SIZE";
        return 1;
    }

    if (argc != 4) {
        std::cerr << "Usage: ./kcliques <graph input file> <k value> <output file>\n";
        return 1;
    }

    int K;
    try {
        K = std::stoi(argv[2]);
    }
    catch(std::exception) {
        std::cerr << "Usage: ./kcliques <graph input file> <k value> <output file>\n";
        return 1;
    }

    std::ifstream input (argv[1]);

    if (input.is_open()) {

        std::string line;
        size_t idx;

        while ( getline (input,line) ) {
            try {
                uint a = std::stoul(line, &idx);
                uint b = std::stoul(&(line[idx]));
                if (a != b) { // Ignorujemy pętle
                    edges.push_back({a, b});
                    degree[a]++;
                    degree[b]++;
                }
            }
            catch(std::exception) {
                std::cerr << "Error: invalid input data format\n";
                input.close();
                return 1;
            }
        }
        input.close();
    }
    else {
        std::cerr << "Error: unable to open input file\n";
        return 1;
    }

    int N = degree.size(); // Liczba wierzchołków

    // Przenumerowanie id na liczby z przedziału od 0 do N
    uint num = 0;
    for (std::unordered_map<uint, uint>::iterator it = degree.begin(); it != degree.end(); ++it) {
        id_to_number[it->first] = num;
        number_to_id[num] = it->first;
        num++;
    }

    // Skierowanie krawędzi
    // TODO: zrobić to na GPU
    for (int i = 0; i < edges.size(); i++) {
        uint a = edges[i].first;
        uint b = edges[i].second;
        if (degree[b] > degree[a] || (degree[b] == degree[a] && id_to_number[b] > id_to_number[a])) {
            edges[i].first = id_to_number[a];
            edges[i].second = id_to_number[b];
        }
        else {
            edges[i].first = id_to_number[b];
            edges[i].second = id_to_number[a];
        }
    }

    // Sortowanie krawędzi wychodzących po numerze sąsiada
    // TODO: zrobić to na GPU
    sort(edges.begin(), edges.end());

    // Sprawdzenie poprawności wejścia
    for (int i = 1; i < edges.size(); ++i) {
        if (edges[i - 1] == edges[i]) {
            std::cerr << "Error: each edge should appear at most once in the list\n";
            return 1;
        }
    }

    int maxDeg = 0;
    // Wyznaczenie przedziału krawędzi wychodzących dla każdego wierzchołka
    std::vector<std::pair<int, int>> intervals(N, {0, 0});
    int l = 0, r = 1;
    while (l < edges.size()) {
        while (r < edges.size() && edges[r].first == edges[l].first) {
            r++;
        }

        intervals[edges[l].first] = {l, r};
        maxDeg = max(maxDeg, r - l);
        l = r;
        r = l + 1;
    }

    maxDeg = ((maxDeg + 31) / 32) * 32; // Zaokrąglam w górę do najbliższej wielokrotności 32
    if (maxDeg > MAX_DEG) {
        std::cerr << "Error: maximal degree after orienting greater than MAX_DEG, may not fit in memory\n";
        return 1;
    }
    int maxStackSize = maxDeg * maxDeg / 2;

    int cliques[K];

    hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

    std::pair<uint, uint>* devEdges;
    std::pair<int, int>* devIntervals;
    HANDLE_ERROR(hipMalloc((void**)&devEdges, sizeof(std::pair<uint, uint>) * edges.size()));
    HANDLE_ERROR(hipMalloc((void**)&devIntervals, sizeof(std::pair<int, int>) * intervals.size()));

    HANDLE_ERROR(hipMemcpy(devEdges, &edges.front(), sizeof(std::pair<uint, uint>) * edges.size(), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(devIntervals, &intervals.front(), sizeof(std::pair<int, int>) * intervals.size(), hipMemcpyHostToDevice));
  
    unsigned int* devIntersect;
    uint* devStackVertex;
    int* devStackDepth;

    int numBlocks = degree.size() < MAX_NUM_BLOCKS ? degree.size() : MAX_NUM_BLOCKS;

    HANDLE_ERROR(hipMalloc((void**)&devIntersect, sizeof(unsigned int) * numBlocks * GROUPS_PER_BLOCK * MAX_DEPTH * (maxDeg / 32)));
    HANDLE_ERROR(hipMalloc((void**)&devStackVertex, sizeof(uint) * maxStackSize * numBlocks * GROUPS_PER_BLOCK));
    HANDLE_ERROR(hipMalloc((void**)&devStackDepth, sizeof(int) * maxStackSize * numBlocks * GROUPS_PER_BLOCK));

    int* devCliques;
    HANDLE_ERROR(hipMalloc((void**)&devCliques, sizeof(int) * numBlocks * GROUPS_PER_BLOCK * K));
    HANDLE_ERROR(hipMemset(devCliques, 0, sizeof(int) * numBlocks * GROUPS_PER_BLOCK * K));

    unsigned int* devInducedSubrgaph;
    HANDLE_ERROR(hipMalloc((void**)&devInducedSubrgaph, sizeof(unsigned int) * numBlocks * maxDeg * (maxDeg / 32)));

    kcliques<<<numBlocks, BLOCK_SIZE>>>(devEdges, devIntervals, N, devIntersect, devStackVertex, devStackDepth, devCliques, K, devInducedSubrgaph, maxDeg, maxStackSize);
    reduce<<<K, numBlocks * GROUPS_PER_BLOCK>>>(devCliques, numBlocks * GROUPS_PER_BLOCK);

    HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Total GPU execution time: %3.1f ms\n", elapsedTime);

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

    HANDLE_ERROR(hipMemcpy(cliques, devCliques, sizeof(int) * K, hipMemcpyDeviceToHost));

    hipFree(devEdges);
    hipFree(devIntervals);
    hipFree(devIntersect);
    hipFree(devStackVertex);
    hipFree(devStackDepth);
    hipFree(devCliques);

    std::ofstream output (argv[3]);
    if (output.is_open()) {
        for (int i = 0; i < K; i++) {
            output << cliques[i] << " ";
        }
        output.close();
    }
    else {
        std::cerr << "Unable to open output file\n";
        return 1;
    }

    return 0;
}
